
#include <hip/hip_runtime.h>
extern "C" __global__ void add(int *a, int *b, int *c) {
    int idx = threadIdx.x;
    c[idx] = a[idx] + b[idx];
}

extern "C" __global__ void multiply(int *a, int *b, int *c) {
    int idx = threadIdx.x;
    c[idx] = a[idx] * b[idx];
}

extern "C" __global__ void dot_product(const int *a, const int *b, int *c, int N) {
    extern __shared__ int cache[];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    int temp = 0;
    while (tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = temp;
    __syncthreads();

    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }

    if (cacheIndex == 0) {
        atomicAdd(c, cache[0]);
    }
}
